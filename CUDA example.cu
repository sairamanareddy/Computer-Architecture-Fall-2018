#define N 512
#define NUM_BLOCKS 16
#define NUM_THREADS 48
//Do not change above three lines.

//Submission should be named as  <RollNumber>_Prog.cu
//Upload just this cu file and nothing else. If you upload it as a zip, it will not be evaluated. 

/*Remember the following guidelines to avoid losing marks
This exercise is quite simple. The only tricky part is that total number of threads (NUM_BLOCKS*NUM_THREADS) may be different (higher or lower) from N.

Index of an array should not exceed the array size. 

No output array-element should be computed more than once

No marks will be given if the program does not compile or run (TAs will not debug your program at all)

Do not change the name of any variable that we have introduced. 
*/



#include <hip/hip_runtime.h>
#include <stdio.h>
#include<stdlib.h>
//TODO: WRITE GPU KERNEL. It should not be called repeatedly from the host, but just once. Each time it is called, it may process more than array-element or not process any array-element at all. 
__global__ void add(int* d_A, int* d_B, int* d_C){
	int tid = NUM_THREADS * blockIdx.x + threadIdx.x;
	while(tid < N){
		for(int j=0;j<N;j++){
		if(tid < N) d_C[tid+j*N]=d_A[tid+j*N]+d_B[tid+j*N];
		}
		tid+=NUM_THREADS*NUM_BLOCKS;
	}
}
int main (int argc, char **argv) {
  int A[N][N], B[N][N], C[N][N];
  int *d_A, *d_B, *d_C; // These are the copies of A, B and C on the GPU
  int *h_C;       // This is a host copy of the output of B from the GPU
  int i, j;
h_C = (int *)malloc(N*N*sizeof(int));	//Allocate memory for the host copy of the output.
  
  for(i=0;i<N;i++) {
    for(j=0;j<N;j++) {
      A[i][j] = i+j;
      B[i][j]= 2*j-1;      
    }
  }

  // sequential implementation of main computation
  for(i=0;i<N;i++) {
    for(j=0;j<N;j++) {
      C[i][j] = A[i][j]+B[i][j];
    }
  }

  // TODO: ALLOCATE MEMORY FOR GPU COPIES OF d_A, d_B and d_C
  hipMalloc((void**)&d_A, N*N*sizeof(int));
  hipMalloc((void**)&d_B, N*N*sizeof(int));
  hipMalloc((void**)&d_C, N*N*sizeof(int));
  // TODO: COPY A TO d_A
    hipMemcpy(d_A, A, N*N*sizeof(int), hipMemcpyHostToDevice);
  // TODO: COPY B TO d_B
    hipMemcpy(d_B, B, N*N*sizeof(int), hipMemcpyHostToDevice);
  // TODO: CREATE BLOCKS with THREADS AND INVOKE GPU KERNEL
   //Use NUM_BLOCKS blocks, each with NUM_THREADS  threads
      add<<<NUM_BLOCKS, NUM_THREADS>>>(d_A, d_B, d_C);
  // TODO: COPY d_C BACK FROM GPU to CPU in variable h_C
      hipMemcpy(h_C, d_C, N*N*sizeof(int), hipMemcpyDeviceToHost);

  // TODO: Verify result is correct by comparing
  for(i=0;i<N;i++) {
    for(j=0;j<N;j++) {
    //TODO: compare each element of h_C and C by subtracting them
        //print only those elements for which the above subtraction is non-zero
	if(h_C[i*N+j]!=C[i][j]){
		printf("ERROR!h_C[%d]=%d!=C[%d][%d]=%d\n", i*N+j, h_C[i*N+j], i, j, C[i][j]);
	}
    }
   }
    //IF even one element of h_C and C differ, report an error.
    //Otherwise, there is no error.
    //If your program is correct, no error should occur.
	free(h_C);hipFree(d_A);hipFree(d_B);hipFree(d_C);
}


